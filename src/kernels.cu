#include "kernels.h"

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// These are the inline versions for all of the SDK helper functions
inline void checkCuBLAS(const hipblasStatus_t error)
{
    if (hipSuccess != error)
    {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error, cublasGetStatusString(error));
        exit(1);
    }
}

inline void check(const hipError_t error)
{
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

void cuda_matmul(int m, int n, int k, const float *A, const float *B, float *C)
{

    // Initialize CUDA and cuBLAS
    hipblasHandle_t handle;
    checkCuBLAS(hipblasCreate(&handle));

    float alpha = 1.0f, beta = 1.0f;

    float *d_A;
    float *d_B;
    float *d_C;

    check(hipMalloc((void **)&d_A, (m * k) * sizeof(float)));
    check(hipMalloc((void **)&d_B, (k * n) * sizeof(float)));
    check(hipMalloc((void **)&d_C, (m * n) * sizeof(float)));

    checkCuBLAS(hipblasSetVector((m * k), sizeof(float), A, 1, d_A, 1));
    checkCuBLAS(hipblasSetVector((k * n), sizeof(float), B, 1, d_B, 1));
    checkCuBLAS(hipblasSetVector((m * n), sizeof(float), C, 1, d_C, 1));

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    checkCuBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m));
    checkCuBLAS(hipblasGetVector(m * n, sizeof(float), d_C, 1, C, 1));

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);
}