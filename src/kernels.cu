#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime.h>

inline void check(const hipError_t error)
{
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

void cuda_dot(float *x, float *y, int n)
{

    float *d_x, *d_y;
    hipError_t err;

    check(hipMalloc(&d_x, n * sizeof(float)));
    hipMalloc(&d_y, n * sizeof(float));

    err = hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("WTF, Why ??? \n");
    }

    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<(n + 255) / 256, 256>>>(n, 10.0f, d_x, d_y);

    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
        printf("%f\n", y[i]);

    hipFree(d_x);
    hipFree(d_y);
    // free(x);
    // free(y);
}